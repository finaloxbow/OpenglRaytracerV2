#include "hip/hip_runtime.h"
#include "ray_collider.h"

#include "../common.h"
#include "../ray/ray.h"

RayCollider::RayCollider(Renderer* renderer)
: renderer(renderer) {}

__device__ float hit_sphere(glm::vec3 center, float radius, Ray* ray) {
	//A = ray origin
	//b = direction
	//c = center

	glm::vec3 oc = ray->origin() - center;
	float a = glm::length2(ray->direction());
	float b = 2.0f * glm::dot(ray->direction(), oc);
	float c = glm::length2(oc) - radius * radius;

	float discriminant = b * b - 4 * a * c;

	if (discriminant < 0)
		return -1;
	else
		//pick closer intersection pt, so minus
		return (-b - sqrt(discriminant)) / (2.0f * a);
}

__device__ glm::vec3 shoot_ray(Ray* ray) {
	float t = hit_sphere(glm::vec3(0, 0, -1), 0.5f, ray);
	if (t > 0.0f) {
		glm::vec3 n = glm::normalize(ray->at(t) - glm::vec3(0, 0, -1));
		return 0.5f * glm::vec3(n.x + 1, n.y + 1, n.z + 1);
	}
	
	glm::vec3 dir = glm::normalize(ray->direction());
	t = (dir.y + 1.0f) / 2.0f;
	return (1.0f - t) * glm::vec3(1, 1, 1) + t * glm::vec3(167.0f / 256.0f, 65.0f / 256.0f, 240 / 256.0f);
}

__global__ void collider_kernel(Renderer* renderer) {
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

	if (row >= WINDOW_HEIGHT || col >= WINDOW_WIDTH) return;

	float viewport_height = 2.0f;
	float viewport_width = viewport_height * ASPECT_RATIO;
	float focal_length = 1.0f;

	glm::vec3 origin(0, 0, 0);
	glm::vec3 horizontal(viewport_width, 0, 0);
	glm::vec3 vertical(0, viewport_height, 0);
	glm::vec3 lower_left_corner = glm::vec3(0, 0, -focal_length) - (horizontal / 2.0f) - (vertical / 2.0f);
	
	float u = float(col) / (WINDOW_WIDTH - 1);
	float v = float(row) / (WINDOW_HEIGHT - 1);

	Ray ray(origin, lower_left_corner + u * horizontal + v * vertical - origin);
	glm::vec3 color = shoot_ray(&ray);
	renderer->update_pixel(row, col, color);
}

void RayCollider::calculate_collisions()
{
	int tx = 8, ty = 8;
	dim3 blocks(WINDOW_HEIGHT / tx + 1, WINDOW_WIDTH / ty + 1);
	dim3 threads(tx, ty);

	collider_kernel<<<blocks, threads>>>(renderer);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

