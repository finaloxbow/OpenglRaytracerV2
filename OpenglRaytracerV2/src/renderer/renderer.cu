#include "renderer.h"

#include <glad/glad.h>
#include <glfw/glfw3.h>

#include <stdexcept>

Renderer::Renderer()
{
    _framebuffer_size = WINDOW_WIDTH * WINDOW_HEIGHT * NUM_CHANNELS;
	hipMallocManaged((void**)&_framebuffer, _framebuffer_size);
	_texture_id = 0;
	_framebuffer_id = 0;

    framebuffer_setup();
}

void Renderer::init()
{
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        throw new std::runtime_error("opengl init failed");
    }
}

void Renderer::framebuffer_setup()
{
	//data alignment stuff
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
	glPixelStorei(GL_UNPACK_ROW_LENGTH, 0);
	glPixelStorei(GL_UNPACK_SKIP_PIXELS, 0);
	glPixelStorei(GL_UNPACK_SKIP_ROWS, 0);

	//texture creation
	glGenTextures(1, &_texture_id);
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, _texture_id);

	glTexImage2D(
		GL_TEXTURE_2D,
		0,
		GL_RGBA,
		WINDOW_WIDTH,
		WINDOW_HEIGHT,
		0,
		GL_RGBA,
		GL_UNSIGNED_BYTE,
		_framebuffer
	);

	// set texture wrapping to GL_REPEAT (default wrapping method)
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	// set texture filtering parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	//generates texture mipmap
	glGenerateMipmap(GL_TEXTURE_2D);

	//creates framebuffer and attaches previous texture to it
	glGenFramebuffers(1, &_framebuffer_id);
	glBindFramebuffer(GL_FRAMEBUFFER, _framebuffer_id);
	glFramebufferTexture(
		GL_FRAMEBUFFER,
		GL_COLOR_ATTACHMENT0,
		_texture_id,
		0
	);

	
}

void Renderer::update_frame()
{
	glTexSubImage2D(
		GL_TEXTURE_2D,
		0,
		0,
		0,
		WINDOW_WIDTH,
		WINDOW_HEIGHT,
		GL_RGBA,
		GL_UNSIGNED_BYTE,
		_framebuffer
	);

	glBindFramebuffer(GL_READ_FRAMEBUFFER, _framebuffer_id);
	glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
	glBlitFramebuffer(
		0, 0, WINDOW_WIDTH, WINDOW_HEIGHT,
		0, 0, WINDOW_WIDTH, WINDOW_HEIGHT,
		GL_COLOR_BUFFER_BIT, GL_NEAREST);
}

__device__ void Renderer::update_pixel(uint64_t x, uint64_t y, glm::vec3& rgba)
{
	if (x < WINDOW_HEIGHT && y < WINDOW_WIDTH) {
		auto red = rgba.x;
		auto green = rgba.y;
		auto blue = rgba.z;

		_framebuffer[x * WINDOW_WIDTH * NUM_CHANNELS + y * NUM_CHANNELS + 0] = static_cast<int>(255.99f * red);
		_framebuffer[x * WINDOW_WIDTH * NUM_CHANNELS + y * NUM_CHANNELS + 1] = static_cast<int>(255.99f * green);
		_framebuffer[x * WINDOW_WIDTH * NUM_CHANNELS + y * NUM_CHANNELS + 2] = static_cast<int>(255.99f * blue);
		_framebuffer[x * WINDOW_WIDTH * NUM_CHANNELS + y * NUM_CHANNELS + 3] = 255;
	}
}
